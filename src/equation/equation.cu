#include "hip/hip_runtime.h"
#include <equation.cuh>







namespace clip {

    Equation::Equation(InputData idata)
    : m_idata(idata), DataArray(idata){




        m_nVelocity = m_idata.nVelocity;





#ifdef ENABLE_2D
        m_ex = new CLIP_INT[WMRT::Q]{0, 1, 0, -1, 0, 1, -1, -1, 1};
        m_ey = new CLIP_INT[WMRT::Q]{0, 0, 1, 0, -1, 1, 1, -1, -1};
        m_wa = new CLIP_REAL[WMRT::Q]{4.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0};

        this->symbolOnDevice(WMRT::ex, m_ex, "ex");
        this->symbolOnDevice(WMRT::ey, m_ey, "ey");
        this->symbolOnDevice(WMRT::wa, m_wa, "wa");

#elif defined(ENABLE_3D)
        m_ex = new CLIP_INT[WMRT::Q]{0, 1, 0, -1, 0, 1, -1, -1, 1};
        m_ey = new CLIP_INT[WMRT::Q]{0, 0, 1, 0, -1, 1, 1, -1, -1};
        m_ez = new CLIP_INT[WMRT::Q]{0, 0, 1, 0, -1, 1, 1, -1, -1};
        m_wa = new CLIP_REAL[WMRT::Q]{4.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0};

        this->symbolOnDevice(WMRT::ex, m_ex, "ex");
        this->symbolOnDevice(WMRT::ey, m_ey, "ey");
        this->symbolOnDevice(WMRT::ez, m_ez, "ez");
        this->symbolOnDevice(WMRT::wa, m_wa, "wa");

#endif



// this->symbolOnDevice(boundary::s_boundaries, m_idata.boundaries.data(), "boundaries");




    }



    Equation::~Equation() {

    if (m_ex)
        delete[] m_ex;
    if (m_ey)
        delete[] m_ey;
    if (m_ez)
        delete[] m_ez;
    if (m_wa)
        delete[] m_wa;

    }



    template <int dof = 1>
    __global__ void periodicBoundary(double* dev_a, double* dev_b = nullptr)
    {
        using namespace boundary;

        const CLIP_UINT i = THREAD_IDX_X;
        const CLIP_UINT j = THREAD_IDX_Y;
        const CLIP_UINT k = (DIM == 3) ? THREAD_IDX_Z : 0;

        const CLIP_UINT idx_SCALAR = DataArray::getIndex(i, j, k);
        const CLIP_UINT idx_X = DataArray::getIndex<DIM>(i, j, k, IDX_X);
        const CLIP_UINT idx_Y = DataArray::getIndex<DIM>(i, j, k, IDX_Y);

#ifdef ENABLE_3D
        const CLIP_UINT idx_Z = DataArray::getIndex<DIM>(i, j, k, IDX_Z);
#endif

//         if (DataArray::isInside<DIM>(i, j, k))
//         {

//             #pragma unroll
//             for (int q = 0; q < dof; ++q)
//             {

//             if(s_boundaries[SideIndex::XMinus].flagCheck(clip::InputBoundary::Type::Periodic)){

//                 dev_a[DataArray::getIndex<dof>(i, j, k, q)] = dev_a[DataArray::getIndex<dof>(i, j, k, q)];
//                 if(dev_b)
//                 dev_b[DataArray::getIndex<dof>(i, j, k, q)] = dev_b[DataArray::getIndex<dof>(i, j, k, q)];
//             }

//             if(s_boundaries[SideIndex::YMinus].flagCheck(clip::InputBoundary::Type::Periodic)){
//                 dev_a[DataArray::getIndex<dof>(i, j, k, q)] = dev_a[DataArray::getIndex<dof>(i, j, k, q)];
//                 if(dev_b)
//                 dev_b[DataArray::getIndex<dof>(i, j, k, q)] = dev_b[DataArray::getIndex<dof>(i, j, k, q)];
//             }

// #ifdef ENABLE_3D
// if(s_boundaries[clip::InputBoundary::Side::ZMinus].flagCheck(clip::InputBoundary::Type::Periodic)){
//     dev_a[DataArray::getIndex<dof>(i, j, k, q)] = dev_a[DataArray::getIndex<dof>(i, j, k, q)];
//     if(dev_b)
//     dev_b[DataArray::getIndex<dof>(i, j, k, q)] = dev_b[DataArray::getIndex<dof>(i, j, k, q)];
// }

// #endif
    

//         }
//         }



    }
    
    template <int Q>
    void Equation::launchPeriodicBoundaryF(
        double* dev_h,
        double* dev_g,
        CLIP_UINT Nx,
        CLIP_UINT Ny,
        CLIP_UINT Nz  // Optional for 2D
    ) {
        dim3 blockDim(16, 16, 1);  // You can adjust for optimal performance
    
    #ifdef ENABLE_3D
        dim3 gridDim(
            (Nx + blockDim.x - 1) / blockDim.x,
            (Ny + blockDim.y - 1) / blockDim.y,
            (Nz + blockDim.z - 1) / blockDim.z);
    #else
        dim3 gridDim(
            (Nx + blockDim.x - 1) / blockDim.x,
            (Ny + blockDim.y - 1) / blockDim.y);
    #endif
    
        // Call the templated kernel
        periodicBoundary<Q><<<gridDim, blockDim>>>(dev_h, dev_g);
    
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        }
    
        hipDeviceSynchronize();  // Optional
    }
    




}

