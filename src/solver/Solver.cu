#include "hip/hip_runtime.h"
#include <Solver.cuh>




namespace clip {




    Solver::Solver(InputData idata)
    : m_idata(idata), DataArray(idata), m_boundary(idata)
{

#ifdef ENABLE_2D


#elif defined(ENABLE_3D)


#endif

    // this->symbolOnDevice(boundary::s_boundaries, m_idata.boundaries.data(), "boundaries");

    // flagGenLauncher3();
}

//     Equation::Equation(InputData idata)
//     : m_idata(idata), DataArray(idata){







//         m_nVelocity = m_idata.nVelocity;





// #ifdef ENABLE_2D
//         m_ex = new CLIP_INT[WMRT::Q]{0, 1, 0, -1, 0, 1, -1, -1, 1};
//         m_ey = new CLIP_INT[WMRT::Q]{0, 0, 1, 0, -1, 1, 1, -1, -1};
//         m_wa = new CLIP_REAL[WMRT::Q]{4.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0};

//         this->symbolOnDevice(WMRT::ex, m_ex, "ex");
//         this->symbolOnDevice(WMRT::ey, m_ey, "ey");
//         this->symbolOnDevice(WMRT::wa, m_wa, "wa");

// #elif defined(ENABLE_3D)
//         m_ex = new CLIP_INT[WMRT::Q]{0, 1, 0, -1, 0, 1, -1, -1, 1};
//         m_ey = new CLIP_INT[WMRT::Q]{0, 0, 1, 0, -1, 1, 1, -1, -1};
//         m_ez = new CLIP_INT[WMRT::Q]{0, 0, 1, 0, -1, 1, 1, -1, -1};
//         m_wa = new CLIP_REAL[WMRT::Q]{4.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0};

//         this->symbolOnDevice(WMRT::ex, m_ex, "ex");
//         this->symbolOnDevice(WMRT::ey, m_ey, "ey");
//         this->symbolOnDevice(WMRT::ez, m_ez, "ez");
//         this->symbolOnDevice(WMRT::wa, m_wa, "wa");

// #endif



// // this->symbolOnDevice(boundary::s_boundaries, m_idata.boundaries.data(), "boundaries");


// flagGenLauncher3();

//     }





    __global__ void flagGen3()
    {
        const CLIP_UINT i = THREAD_IDX_X;
        const CLIP_UINT j = THREAD_IDX_Y;
        const CLIP_UINT k = (DIM == 3) ? THREAD_IDX_Z : 0;

        // const CLIP_UINT idx_SCALAR = DataArray::getIndex(i, j, k);

        // printf("Thread index: i = %d, j = %d, k = %d\n", i, j, k);
        // printf("Thread index2: %d \n", idx_SCALAR);

        // if (DataArray::isInside<DIM>(i, j, k)){

            // printf("Thread index: i = %d, j = %d, k = %d\n", i, j, k);
            // printf("index: i = %d\n", DataArray::getDomainExtent(IDX_X));
            // printf("index: s_domainExtent = %d\n", s_domainExtent[IDX_Y]);

        // }
        // printf("index: inside equation getDomainExtent = %d\n", DataArray::getDomainExtent(1));
        // printf("index:  inside equation s_domainExtent = %d\n", s_domainExtent[IDX_Y]);

        // printf("index:  inside equation ex = %d\n", WMRT::ex[3]);
        // printf("index: i = %d\n", idx_SCALAR);
            // printf("Thread index: i = %d, j = %d, k = %d\n", i, j, k);
    }



    void Solver::flagGenLauncher3 (){

             flagGen3<<<dimGrid, dimBlock>>>();
             hipDeviceSynchronize();  
    }






    Solver::~Solver() {

    if (m_ex)
        delete[] m_ex;
    if (m_ey)
        delete[] m_ey;
    if (m_ez)
        delete[] m_ez;
    if (m_wa)
        delete[] m_wa;

    }



    template <int dof = 1>
    __global__ void periodicBoundary(double* dev_a, double* dev_b = nullptr)
    {
        using namespace boundary;

        const CLIP_UINT i = THREAD_IDX_X;
        const CLIP_UINT j = THREAD_IDX_Y;
        const CLIP_UINT k = (DIM == 3) ? THREAD_IDX_Z : 0;

        // const CLIP_UINT idx_SCALAR = DataArray::getIndex(i, j, k);
        // const CLIP_UINT idx_X = DataArray::getIndex<DIM>(i, j, k, IDX_X);
        // const CLIP_UINT idx_Y = DataArray::getIndex<DIM>(i, j, k, IDX_Y);

#ifdef ENABLE_3D
        const CLIP_UINT idx_Z = DataArray::getIndex<DIM>(i, j, k, IDX_Z);
#endif

//         if (DataArray::isInside<DIM>(i, j, k))
//         {

//             #pragma unroll
//             for (int q = 0; q < dof; ++q)
//             {

//             if(s_boundaries[SideIndex::XMinus].flagCheck(clip::InputBoundary::Type::Periodic)){

//                 dev_a[DataArray::getIndex<dof>(i, j, k, q)] = dev_a[DataArray::getIndex<dof>(i, j, k, q)];
//                 if(dev_b)
//                 dev_b[DataArray::getIndex<dof>(i, j, k, q)] = dev_b[DataArray::getIndex<dof>(i, j, k, q)];
//             }

//             if(s_boundaries[SideIndex::YMinus].flagCheck(clip::InputBoundary::Type::Periodic)){
//                 dev_a[DataArray::getIndex<dof>(i, j, k, q)] = dev_a[DataArray::getIndex<dof>(i, j, k, q)];
//                 if(dev_b)
//                 dev_b[DataArray::getIndex<dof>(i, j, k, q)] = dev_b[DataArray::getIndex<dof>(i, j, k, q)];
//             }

// #ifdef ENABLE_3D
// if(s_boundaries[clip::InputBoundary::Side::ZMinus].flagCheck(clip::InputBoundary::Type::Periodic)){
//     dev_a[DataArray::getIndex<dof>(i, j, k, q)] = dev_a[DataArray::getIndex<dof>(i, j, k, q)];
//     if(dev_b)
//     dev_b[DataArray::getIndex<dof>(i, j, k, q)] = dev_b[DataArray::getIndex<dof>(i, j, k, q)];
// }

// #endif
    

//         }
//         }



    }
    
    template <int Q>
    void Solver::launchPeriodicBoundaryF(
        double* dev_h,
        double* dev_g,
        CLIP_UINT Nx,
        CLIP_UINT Ny,
        CLIP_UINT Nz  // Optional for 2D
    ) {
        dim3 blockDim(16, 16, 1);  // You can adjust for optimal performance
    
    #ifdef ENABLE_3D
        dim3 gridDim(
            (Nx + blockDim.x - 1) / blockDim.x,
            (Ny + blockDim.y - 1) / blockDim.y,
            (Nz + blockDim.z - 1) / blockDim.z);
    #else
        dim3 gridDim(
            (Nx + blockDim.x - 1) / blockDim.x,
            (Ny + blockDim.y - 1) / blockDim.y);
    #endif
    
        // Call the templated kernel
        periodicBoundary<Q><<<gridDim, blockDim>>>(dev_h, dev_g);
    
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        }
    
        hipDeviceSynchronize();  // Optional
    }
    




}

