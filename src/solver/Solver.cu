#include "hip/hip_runtime.h"
#include <Solver.cuh>

namespace clip
{

    Solver::Solver(const InputData &idata, const Domain &domain, DataArray &DA, const Boundary &boundary, const Geometry &geom)
        : m_idata(&idata), m_domain(&domain), m_DA(&DA), m_boundary(&boundary), m_geom(&geom)
    {

        dimGrid = m_DA->dimGrid;
        dimBlock = m_DA->dimBlock;
        m_info = m_domain->info;
        m_BCMap = m_boundary->BCMap;
        m_geomPool = m_geom->getDeviceStruct();

#ifdef ENABLE_2D

#elif defined(ENABLE_3D)

#endif
    }

    Solver::~Solver()
    {
    }

    template <int dof = 1>
    __global__ void kernelPeriodicBoundary(const Domain::DomainInfo domain, const Boundary::BCTypeMap BCmap,
                                           CLIP_REAL *dev_a, CLIP_REAL *dev_b = nullptr)
    {

        const CLIP_UINT i = THREAD_IDX_X;
        const CLIP_UINT j = THREAD_IDX_Y;
        const CLIP_UINT k = (DIM == 3) ? THREAD_IDX_Z : 0;

        const CLIP_UINT idx_SCALAR = Domain::getIndex(domain, i, j, k);

        // printf("i: %d,    j: %d", i, j);

        if (Domain::isInside<DIM>(domain, i, j, k))
        {

            // printf("i = %/d,  j = %d \n", i,j);
#pragma unroll
            for (CLIP_UINT q = 0; q < dof; ++q)
            {

                if (BCmap.types[object::XMinus] == Boundary::Type::Periodic || BCmap.types[object::XPlus] == Boundary::Type::Periodic)
                {
                    dev_a[Domain::getIndex<dof>(domain, domain.ghostDomainMinIdx[IDX_X], j, k, q)] = dev_a[Domain::getIndex<dof>(domain, domain.domainMaxIdx[IDX_X], j, k, q)];
                    dev_a[Domain::getIndex<dof>(domain, domain.ghostDomainMaxIdx[IDX_X], j, k, q)] = dev_a[Domain::getIndex<dof>(domain, domain.domainMinIdx[IDX_X], j, k, q)];

                    // printf("i: %d,    j: %d", i, j);

                    if (dev_b)
                    {
                        dev_b[Domain::getIndex<dof>(domain, domain.ghostDomainMinIdx[IDX_X], j, k, q)] = dev_b[Domain::getIndex<dof>(domain, domain.domainMaxIdx[IDX_X], j, k, q)];
                        dev_b[Domain::getIndex<dof>(domain, domain.ghostDomainMaxIdx[IDX_X], j, k, q)] = dev_b[Domain::getIndex<dof>(domain, domain.domainMinIdx[IDX_X], j, k, q)];
                    }
                }

                if (BCmap.types[object::YMinus] == Boundary::Type::Periodic || BCmap.types[object::YPlus] == Boundary::Type::Periodic)
                {

                    dev_a[Domain::getIndex<dof>(domain, i, domain.ghostDomainMinIdx[IDX_Y], k, q)] = dev_a[Domain::getIndex<dof>(domain, i, domain.domainMaxIdx[IDX_Y], k, q)];
                    dev_a[Domain::getIndex<dof>(domain, i, domain.ghostDomainMaxIdx[IDX_Y], k, q)] = dev_a[Domain::getIndex<dof>(domain, i, domain.domainMinIdx[IDX_Y], k, q)];

                    if (dev_b)
                    {
                        dev_b[Domain::getIndex<dof>(domain, i, domain.ghostDomainMinIdx[IDX_Y], k, q)] = dev_b[Domain::getIndex<dof>(domain, i, domain.domainMaxIdx[IDX_Y], k, q)];
                        dev_b[Domain::getIndex<dof>(domain, i, domain.ghostDomainMaxIdx[IDX_Y], k, q)] = dev_b[Domain::getIndex<dof>(domain, i, domain.domainMinIdx[IDX_Y], k, q)];
                    }
                }

#ifdef ENABLE_3D

                if (BCmap.types[object::ZMinus] == Boundary::Type::Periodic || BCmap.types[object::ZPlus] == Boundary::Type::Periodic)
                {

                    dev_a[Domain::getIndex<dof>(domain, i, j, domain.ghostDomainMinIdx[IDX_Z], q)] = dev_a[Domain::getIndex<dof>(domain, i, j, domain.domainMaxIdx[IDX_Z], q)];
                    dev_a[Domain::getIndex<dof>(domain, i, j, domain.ghostDomainMaxIdx[IDX_Z], q)] = dev_a[Domain::getIndex<dof>(domain, i, j, domain.domainMinIdx[IDX_Z], q)];

                    if (dev_b)
                    {
                        dev_b[Domain::getIndex<dof>(domain, i, j, domain.ghostDomainMinIdx[IDX_Z], q)] = dev_b[Domain::getIndex<dof>(domain, i, j, domain.domainMaxIdx[IDX_Z], q)];
                        dev_b[Domain::getIndex<dof>(domain, i, j, domain.ghostDomainMaxIdx[IDX_Z], q)] = dev_b[Domain::getIndex<dof>(domain, i, j, domain.domainMinIdx[IDX_Z], q)];
                    }
                }

#endif
            }
        }
    }

    template <CLIP_UINT Q, CLIP_UINT dof>
    __global__ void kernelFullBouncBack(const Domain::DomainInfo domain, const Boundary::BCTypeMap BCmap, const WMRT::wallBCMap wallBCMap,
                                        CLIP_REAL *dev_a, CLIP_REAL *dev_b = nullptr)
    {

        const CLIP_UINT i = THREAD_IDX_X;
        const CLIP_UINT j = THREAD_IDX_Y;
        const CLIP_UINT k = (DIM == 3) ? THREAD_IDX_Z : 0;

        const CLIP_UINT idx_SCALAR = Domain::getIndex(domain, i, j, k);

        // printf("i: %d,    j: %d", i, j);

        if (Domain::isInside<DIM>(domain, i, j, k))
        {

            // printf("i = %/d,  j = %d \n", i,j);
#pragma unroll
            for (CLIP_UINT q = 0; q < dof; ++q)
            {

                if (BCmap.types[object::XMinus] == Boundary::Type::Wall)
                {
                    dev_a[Domain::getIndex<Q>(domain, domain.ghostDomainMinIdx[IDX_X], j, k, wallBCMap.XMinus[q])] = dev_a[Domain::getIndex<Q>(domain, domain.domainMinIdx[IDX_X], j, k, wallBCMap.XPlus[q])];

                    if (dev_b)
                    {
                        dev_b[Domain::getIndex<Q>(domain, domain.ghostDomainMinIdx[IDX_X], j, k, wallBCMap.XMinus[q])] = dev_b[Domain::getIndex<Q>(domain, domain.domainMinIdx[IDX_X], j, k, wallBCMap.XPlus[q])];
                    }
                }

                if (BCmap.types[object::XPlus] == Boundary::Type::Wall)
                {
                    dev_a[Domain::getIndex<Q>(domain, domain.ghostDomainMaxIdx[IDX_X], j, k, wallBCMap.XPlus[q])] = dev_a[Domain::getIndex<Q>(domain, domain.domainMaxIdx[IDX_X], j, k, wallBCMap.XMinus[q])];

                    if (dev_b)
                    {
                        dev_b[Domain::getIndex<Q>(domain, domain.ghostDomainMaxIdx[IDX_X], j, k, wallBCMap.XPlus[q])] = dev_b[Domain::getIndex<Q>(domain, domain.domainMaxIdx[IDX_X], j, k, wallBCMap.XMinus[q])];
                    }
                }

                if (BCmap.types[object::YMinus] == Boundary::Type::Wall)
                {

                    dev_a[Domain::getIndex<Q>(domain, i, domain.ghostDomainMinIdx[IDX_Y], k, wallBCMap.YMinus[q])] = dev_a[Domain::getIndex<Q>(domain, i, domain.domainMinIdx[IDX_Y], k, wallBCMap.XPlus[q])];

                    if (dev_b)
                    {
                        dev_b[Domain::getIndex<Q>(domain, i, domain.ghostDomainMinIdx[IDX_Y], k, wallBCMap.YMinus[q])] = dev_b[Domain::getIndex<Q>(domain, i, domain.domainMinIdx[IDX_Y], k, wallBCMap.XPlus[q])];
                    }
                }

                if (BCmap.types[object::YPlus] == Boundary::Type::Wall)
                {

                    dev_a[Domain::getIndex<Q>(domain, i, domain.ghostDomainMaxIdx[IDX_Y], k, wallBCMap.YPlus[q])] = dev_a[Domain::getIndex<Q>(domain, i, domain.domainMaxIdx[IDX_Y], k, wallBCMap.YMinus[q])];

                    if (dev_b)
                    {
                        dev_b[Domain::getIndex<Q>(domain, i, domain.ghostDomainMaxIdx[IDX_Y], k, wallBCMap.YPlus[q])] = dev_b[Domain::getIndex<Q>(domain, i, domain.domainMaxIdx[IDX_Y], k, wallBCMap.YMinus[q])];
                    }
                }

#ifdef ENABLE_3D

                if (BCmap.types[object::ZMinus] == Boundary::Type::Wall)
                {

                    dev_a[Domain::getIndex<Q>(domain, i, j, domain.ghostDomainMinIdx[IDX_Z], wallBCMap.ZMinus[q])] = dev_a[Domain::getIndex<Q>(domain, i, j, domain.domainMinIdx[IDX_Z], wallBCMap.ZPlus[q])];

                    if (dev_b)
                    {
                        dev_b[Domain::getIndex<Q>(domain, i, j, domain.ghostDomainMinIdx[IDX_Z], wallBCMap.ZMinus[q])] = dev_b[Domain::getIndex<Q>(domain, i, j, domain.domainMinIdx[IDX_Z], wallBCMap.ZPlus[q])];
                    }
                }

                if (BCmap.types[object::ZPlus] == Boundary::Type::Wall)
                {

                    dev_a[Domain::getIndex<Q>(domain, i, j, domain.ghostDomainMaxIdx[IDX_Z], wallBCMap.ZPlus[q])] = dev_a[Domain::getIndex<Q>(domain, i, j, domain.domainMaxIdx[IDX_Z], wallBCMap.ZMinus[q])];

                    if (dev_b)
                    {
                        dev_b[Domain::getIndex<Q>(domain, i, j, domain.ghostDomainMaxIdx[IDX_Z], wallBCMap.ZPlus[q])] = dev_b[Domain::getIndex<Q>(domain, i, j, domain.domainMaxIdx[IDX_Z], wallBCMap.ZMinus[q])];
                    }
                }

#endif
            }
        }
    }

    template <CLIP_UINT Q, CLIP_UINT dof, typename T>
    __global__ void kernelHalfBounceBack(const Domain::DomainInfo domain, const Boundary::BCTypeMap BCmap,
                                         const T wallMap, CLIP_REAL *dev_a, CLIP_REAL *dev_a_post, CLIP_REAL *dev_b, CLIP_REAL *dev_b_post)
    {
        const CLIP_UINT i = THREAD_IDX_X;
        const CLIP_UINT j = THREAD_IDX_Y;
        const CLIP_UINT k = (DIM == 3) ? THREAD_IDX_Z : 0;

        if (Domain::isInside<DIM>(domain, i, j, k))
        {
#pragma unroll
            for (CLIP_UINT q = 0; q < dof; ++q)
            {
                // X boundaries
                if ((BCmap.types[object::XMinus] == Boundary::Type::Wall ||
                     BCmap.types[object::XMinus] == Boundary::Type::SlipWall) &&
                    i == domain.domainMinIdx[IDX_X])
                {
                    const CLIP_UINT idx = Domain::getIndex<Q>(domain, i, j, k, wallMap.XMinus[q]);
                    const CLIP_UINT opp_idx = Domain::getIndex<Q>(domain, i, j, k, wallMap.XPlus[q]);
                    dev_a_post[idx] = dev_a[opp_idx];
                    if (dev_b)
                        dev_b_post[idx] = dev_b[opp_idx];
                }

                if ((BCmap.types[object::XPlus] == Boundary::Type::Wall ||
                     BCmap.types[object::XPlus] == Boundary::Type::SlipWall) &&
                    i == domain.domainMaxIdx[IDX_X])
                {
                    const CLIP_UINT idx = Domain::getIndex<Q>(domain, i, j, k, wallMap.XPlus[q]);
                    const CLIP_UINT opp_idx = Domain::getIndex<Q>(domain, i, j, k, wallMap.XMinus[q]);
                    dev_a_post[idx] = dev_a[opp_idx];
                    if (dev_b)
                        dev_b_post[idx] = dev_b[opp_idx];
                }

                // Y boundaries
                if ((BCmap.types[object::YMinus] == Boundary::Type::Wall ||
                     BCmap.types[object::YMinus] == Boundary::Type::SlipWall) &&
                    j == domain.domainMinIdx[IDX_Y])
                {
                    const CLIP_UINT idx = Domain::getIndex<Q>(domain, i, j, k, wallMap.YMinus[q]);
                    const CLIP_UINT opp_idx = Domain::getIndex<Q>(domain, i, j, k, wallMap.YPlus[q]);
                    dev_a_post[idx] = dev_a[opp_idx];
                    if (dev_b)
                        dev_b_post[idx] = dev_b[opp_idx];
                }

                if ((BCmap.types[object::YPlus] == Boundary::Type::Wall ||
                     BCmap.types[object::YPlus] == Boundary::Type::SlipWall) &&
                    j == domain.domainMaxIdx[IDX_Y])
                {
                    const CLIP_UINT idx = Domain::getIndex<Q>(domain, i, j, k, wallMap.YPlus[q]);
                    const CLIP_UINT opp_idx = Domain::getIndex<Q>(domain, i, j, k, wallMap.YMinus[q]);
                    dev_a_post[idx] = dev_a[opp_idx];
                    if (dev_b)
                        dev_b_post[idx] = dev_b[opp_idx];
                }

#ifdef ENABLE_3D
                // Z boundaries
                if ((BCmap.types[object::ZMinus] == Boundary::Type::Wall ||
                     BCmap.types[object::ZMinus] == Boundary::Type::SlipWall) &&
                    k == domain.domainMinIdx[IDX_Z])
                {
                    const CLIP_UINT idx = Domain::getIndex<Q>(domain, i, j, k, wallMap.ZMinus[q]);
                    const CLIP_UINT opp_idx = Domain::getIndex<Q>(domain, i, j, k, wallMap.ZPlus[q]);
                    dev_a_post[idx] = dev_a[opp_idx];
                    if (dev_b)
                        dev_b_post[idx] = dev_b[opp_idx];
                }

                if ((BCmap.types[object::ZPlus] == Boundary::Type::Wall ||
                     BCmap.types[object::ZPlus] == Boundary::Type::SlipWall) &&
                    k == domain.domainMaxIdx[IDX_Z])
                {
                    const CLIP_UINT idx = Domain::getIndex<Q>(domain, i, j, k, wallMap.ZPlus[q]);
                    const CLIP_UINT opp_idx = Domain::getIndex<Q>(domain, i, j, k, wallMap.ZMinus[q]);
                    dev_a_post[idx] = dev_a[opp_idx];
                    if (dev_b)
                        dev_b_post[idx] = dev_b[opp_idx];
                }
#endif
            }
        }
    }

    template <CLIP_UINT Q, CLIP_UINT dof, typename T>
    __global__ void kernelFreeConvect(const Domain::DomainInfo domain, const Boundary::BCTypeMap BCmap,
                                      const T wallMap, CLIP_REAL *dev_vel,
                                      CLIP_REAL *dev_a, CLIP_REAL *dev_a_prev,
                                      CLIP_REAL *dev_b, CLIP_REAL *dev_b_prev)
    {
        const CLIP_UINT i = THREAD_IDX_X;
        const CLIP_UINT j = THREAD_IDX_Y;
        const CLIP_UINT k = (DIM == 3) ? THREAD_IDX_Z : 0;

        if (Domain::isInside<DIM>(domain, i, j, k))
        {

            // XMIN
            if (BCmap.types[object::XMinus] == Boundary::Type::FreeConvect && i == domain.domainMinIdx[IDX_X])
            {
#pragma unroll
                for (CLIP_UINT q = 0; q < dof; ++q)
                {
                    const CLIP_UINT idxBoundary = Domain::getIndex<Q>(domain, i, j, k, wallMap.XMinus[q]);
                    const CLIP_UINT idxInterior = Domain::getIndex<Q>(domain, i + 1, j, k, wallMap.XMinus[q]);
                    const CLIP_REAL convectVel = fabs(dev_vel[Domain::getIndex<DIM>(domain, i + 1, j, k, IDX_X)]);

                    dev_a[idxBoundary] = (dev_a_prev[idxBoundary] + convectVel * dev_a[idxInterior]) / (1.0 + convectVel);
                    if (dev_b)
                        dev_b[idxBoundary] = (dev_b_prev[idxBoundary] + convectVel * dev_b[idxInterior]) / (1.0 + convectVel);
                }
            }

            // XMAX
            if (BCmap.types[object::XPlus] == Boundary::Type::FreeConvect && i == domain.domainMaxIdx[IDX_X])
            {
#pragma unroll
                for (CLIP_UINT q = 0; q < dof; ++q)
                {
                    const CLIP_UINT idxBoundary = Domain::getIndex<Q>(domain, i, j, k, wallMap.XPlus[q]);
                    const CLIP_UINT idxInterior = Domain::getIndex<Q>(domain, i - 1, j, k, wallMap.XPlus[q]);
                    const CLIP_REAL convectVel = fabs(dev_vel[Domain::getIndex<DIM>(domain, i - 1, j, k, IDX_X)]);

                    dev_a[idxBoundary] = (dev_a_prev[idxBoundary] + convectVel * dev_a[idxInterior]) / (1.0 + convectVel);
                    if (dev_b)
                        dev_b[idxBoundary] = (dev_b_prev[idxBoundary] + convectVel * dev_b[idxInterior]) / (1.0 + convectVel);
                }
            }

            // YMIN
            if (BCmap.types[object::YMinus] == Boundary::Type::FreeConvect && j == domain.domainMinIdx[IDX_Y])
            {
#pragma unroll
                for (CLIP_UINT q = 0; q < dof; ++q)
                {
                    const CLIP_UINT idxBoundary = Domain::getIndex<Q>(domain, i, j, k, wallMap.YMinus[q]);
                    const CLIP_UINT idxInterior = Domain::getIndex<Q>(domain, i, j + 1, k, wallMap.YMinus[q]);
                    const CLIP_REAL convectVel = fabs(dev_vel[Domain::getIndex<DIM>(domain, i, j + 1, k, IDX_Y)]);

                    dev_a[idxBoundary] = (dev_a_prev[idxBoundary] + convectVel * dev_a[idxInterior]) / (1.0 + convectVel);
                    if (dev_b)
                        dev_b[idxBoundary] = (dev_b_prev[idxBoundary] + convectVel * dev_b[idxInterior]) / (1.0 + convectVel);
                }
            }

            // YMAX
            if (BCmap.types[object::YPlus] == Boundary::Type::FreeConvect && j == domain.domainMaxIdx[IDX_Y])
            {
#pragma unroll
                for (CLIP_UINT q = 0; q < dof; ++q)
                {
                    const CLIP_UINT idxBoundary = Domain::getIndex<Q>(domain, i, j, k, wallMap.YPlus[q]);
                    const CLIP_UINT idxInterior = Domain::getIndex<Q>(domain, i, j - 1, k, wallMap.YPlus[q]);
                    const CLIP_REAL convectVel = fabs(dev_vel[Domain::getIndex<DIM>(domain, i, j - 1, k, IDX_Y)]);

                    dev_a[idxBoundary] = (dev_a_prev[idxBoundary] + convectVel * dev_a[idxInterior]) / (1.0 + convectVel);
                    if (dev_b)
                        dev_b[idxBoundary] = (dev_b_prev[idxBoundary] + convectVel * dev_b[idxInterior]) / (1.0 + convectVel);
                }
            }

#ifdef ENABLE_3D
            // ZMIN
            if (BCmap.types[object::ZMinus] == Boundary::Type::FreeConvect && k == domain.domainMinIdx[IDX_Z])
            {
#pragma unroll
                for (CLIP_UINT q = 0; q < dof; ++q)
                {
                    const CLIP_UINT idxBoundary = Domain::getIndex<Q>(domain, i, j, k, wallMap.ZMinus[q]);
                    const CLIP_UINT idxInterior = Domain::getIndex<Q>(domain, i, j, k + 1, wallMap.ZMinus[q]);
                    const CLIP_REAL convectVel = fabs(dev_vel[Domain::getIndex<DIM>(domain, i, j, k + 1, IDX_Z)]);

                    dev_a[idxBoundary] = (dev_a_prev[idxBoundary] + convectVel * dev_a[idxInterior]) / (1.0 + convectVel);
                    if (dev_b)
                        dev_b[idxBoundary] = (dev_b_prev[idxBoundary] + convectVel * dev_b[idxInterior]) / (1.0 + convectVel);
                }
            }

            // ZMAX
            if (BCmap.types[object::ZPlus] == Boundary::Type::FreeConvect && k == domain.domainMaxIdx[IDX_Z])
            {
#pragma unroll
                for (CLIP_UINT q = 0; q < dof; ++q)
                {
                    const CLIP_UINT idxBoundary = Domain::getIndex<Q>(domain, i, j, k, wallMap.ZPlus[q]);
                    const CLIP_UINT idxInterior = Domain::getIndex<Q>(domain, i, j, k - 1, wallMap.ZPlus[q]);
                    const CLIP_REAL convectVel = fabs(dev_vel[Domain::getIndex<DIM>(domain, i, j, k - 1, IDX_Z)]);

                    dev_a[idxBoundary] = (dev_a_prev[idxBoundary] + convectVel * dev_a[idxInterior]) / (1.0 + convectVel);
                    if (dev_b)
                        dev_b[idxBoundary] = (dev_b_prev[idxBoundary] + convectVel * dev_b[idxInterior]) / (1.0 + convectVel);
                }
            }
#endif
        }
    }

    template <CLIP_UINT Q, CLIP_UINT dof, typename T>
    __global__ void kernelNeumann(const Domain::DomainInfo domain, const Boundary::BCTypeMap BCmap,
                                  const T wallMap, CLIP_REAL *dev_a, CLIP_REAL *dev_b)
    {
        const CLIP_UINT i = THREAD_IDX_X;
        const CLIP_UINT j = THREAD_IDX_Y;
        const CLIP_UINT k = (DIM == 3) ? THREAD_IDX_Z : 0;

        if (Domain::isInside<DIM>(domain, i, j, k))
        {
#pragma unroll
            for (CLIP_UINT q = 0; q < dof; ++q)
            {

                // X boundaries
                if (BCmap.types[object::XMinus] == Boundary::Type::Neumann && i == domain.domainMinIdx[IDX_X])
                {
                    const CLIP_UINT idxBoundary = Domain::getIndex<Q>(domain, i, j, k, wallMap.XMinus[q]);
                    const CLIP_UINT idxInterior = Domain::getIndex<Q>(domain, i + 1, j, k, wallMap.XMinus[q]);

                    dev_a[idxBoundary] = dev_a[idxInterior];
                    if (dev_b)
                        dev_b[idxBoundary] = dev_b[idxInterior];
                }

                if (BCmap.types[object::XPlus] == Boundary::Type::Neumann && i == domain.domainMaxIdx[IDX_X])
                {
                    const CLIP_UINT idxBoundary = Domain::getIndex<Q>(domain, i, j, k, wallMap.XPlus[q]);
                    const CLIP_UINT idxInterior = Domain::getIndex<Q>(domain, i - 1, j, k, wallMap.XPlus[q]);

                    dev_a[idxBoundary] = dev_a[idxInterior];
                    if (dev_b)
                        dev_b[idxBoundary] = dev_b[idxInterior];
                }

                // Y boundaries
                if (BCmap.types[object::YMinus] == Boundary::Type::Neumann && j == domain.domainMinIdx[IDX_Y])
                {
                    const CLIP_UINT idxBoundary = Domain::getIndex<Q>(domain, i, j, k, wallMap.YMinus[q]);
                    const CLIP_UINT idxInterior = Domain::getIndex<Q>(domain, i, j + 1, k, wallMap.YMinus[q]);

                    dev_a[idxBoundary] = dev_a[idxInterior];
                    if (dev_b)
                        dev_b[idxBoundary] = dev_b[idxInterior];
                }

                if (BCmap.types[object::YPlus] == Boundary::Type::Neumann && j == domain.domainMaxIdx[IDX_Y])
                {
                    const CLIP_UINT idxBoundary = Domain::getIndex<Q>(domain, i, j, k, wallMap.YPlus[q]);
                    const CLIP_UINT idxInterior = Domain::getIndex<Q>(domain, i, j - 1, k, wallMap.YPlus[q]);

                    dev_a[idxBoundary] = dev_a[idxInterior];
                    if (dev_b)
                        dev_b[idxBoundary] = dev_b[idxInterior];
                }

#ifdef ENABLE_3D
                // Z boundaries
                if (BCmap.types[object::ZMinus] == Boundary::Type::Neumann && k == domain.domainMinIdx[IDX_Z])
                {
                    const CLIP_UINT idxBoundary = Domain::getIndex<Q>(domain, i, j, k, wallMap.ZMinus[q]);
                    const CLIP_UINT idxInterior = Domain::getIndex<Q>(domain, i, j, k + 1, wallMap.ZMinus[q]);

                    dev_a[idxBoundary] = dev_a[idxInterior];
                    if (dev_b)
                        dev_b[idxBoundary] = dev_b[idxInterior];
                }

                if (BCmap.types[object::ZPlus] == Boundary::Type::Neumann && k == domain.domainMaxIdx[IDX_Z])
                {
                    const CLIP_UINT idxBoundary = Domain::getIndex<Q>(domain, i, j, k, wallMap.ZPlus[q]);
                    const CLIP_UINT idxInterior = Domain::getIndex<Q>(domain, i, j, k - 1, wallMap.ZPlus[q]);

                    dev_a[idxBoundary] = dev_a[idxInterior];
                    if (dev_b)
                        dev_b[idxBoundary] = dev_b[idxInterior];
                }
#endif
            }
        }
    }

    /////////////////////////////////////////////////////// start

    // template <typname T, typename Q, typename R, typename S>
    __global__ void JetBoundary(const Domain::DomainInfo domain, const Geometry::GeometryDevice geom, const Boundary::BCTypeMap BCmap,
                                const WMRT::WMRTvelSet velSet, const WMRT::slipWallBCMap wallBCMap, CLIP_REAL *dev_c, CLIP_REAL *dev_f, CLIP_REAL *dev_g)
    {
        // const WMRT::velocityBCMap velocityBCMap
        float feq[19], geq[19];
        // int FStop[10] = {4, 8, 9, 16, 18, 3, 10, 7, 17, 15};


        const CLIP_UINT Q = velSet.Q;


        const CLIP_UINT i = THREAD_IDX_X;
        const CLIP_UINT j = THREAD_IDX_Y;
        const CLIP_UINT k = (DIM == 3) ? THREAD_IDX_Z : 0;

        const CLIP_UINT idx_SCALAR = Domain::getIndex(domain, i, j, k);

        const CLIP_REAL x = static_cast<CLIP_REAL>(i);
        const CLIP_REAL y = static_cast<CLIP_REAL>(j);
        const CLIP_REAL z = (DIM == 3) ? static_cast<CLIP_REAL>(k) : 0.0;


        if (Domain::isInside<DIM>(domain, i, j, k))
        
        {


            // if (BCmap.types[object::YPlus] == Boundary::Type::Velocity && i == domain.ghostDomainMaxIdx[IDX_X])
            if (j == domain.ghostDomainMaxIdx[IDX_Y])
            {

                if (Geometry::sdf(geom, 0, x, y, z) <= 0)
                {

                    CLIP_REAL Mx = 0;
                    CLIP_REAL Mz = 0;
                    CLIP_REAL N = 0;
#pragma unroll
                    for (int q = 1; q < 19; q++)
                    {

                        const CLIP_REAL fa_wa = Solver::Equilibrium_new(velSet, q, BCmap.val[object::YPlus][IDX_X],
                                                                         BCmap.val[object::YPlus][IDX_Y], BCmap.val[object::YPlus][IDX_Z]);

                                                                         
                        feq[q] = 0.0 * velSet.wa[q] + fa_wa;
                        geq[q] = dev_c[idx_SCALAR] * (fa_wa + velSet.wa[q]);

                        if (velSet.ey[q] == 0)
                        {
                            Mx += velSet.ex[q] * (dev_f[Domain::getIndex<Q>(domain, i, j, k, q)] - feq[q]);
                            N += (dev_g[Domain::getIndex<Q>(domain, i, j, k, q)] - geq[q]);
                        }

                        if (velSet.ex[q] == 0)
                            Mz += velSet.ez[q] * (dev_f[Domain::getIndex<Q>(domain, i, j, k, q)] - feq[q]);
                    }

                    for (int q = 0; q < 5; q++)
                    {

                        const CLIP_UINT idxF = Domain::getIndex<Q>(domain, i, j, k, wallBCMap.YPlus[q]);
                        const CLIP_UINT oppos_idxF = Domain::getIndex<Q>(domain, i, j, k, wallBCMap.YMinus[q]);
                        const CLIP_UINT idx = wallBCMap.YPlus[q];
                        const CLIP_UINT oppos_idx = wallBCMap.YMinus[q];

                        dev_f[idxF] = dev_f[oppos_idxF] + feq[idx] - feq[oppos_idx] - (1.0 / 4.0) * (velSet.ex[idx] * Mx + velSet.ez[idx] * Mz);

                        // dev_g[getIndexf(i, j, k, 4)] = geq[4] + dev_g[getIndexf(i, j, k, 3)] - geq[3];
                        // dev_g[getIndexf(i, j, k, 8)] = dev_g[getIndexf(i, j, k, 7)] + geq[8] - geq[7] - (1.0 / 4.0) * (ex[8] * Mx + ez[8] * Mz);
                        // dev_g[getIndexf(i, j, k, 9)] = dev_g[getIndexf(i, j, k, 10)] + geq[9] - geq[10] - (1.0 / 4.0) * (ex[9] * Mx + ez[9] * Mz);
                        // dev_g[getIndexf(i, j, k, 16)] = dev_g[getIndexf(i, j, k, 15)] + geq[16] - geq[15] - (1.0 / 4.0) * (ex[16] * Mx + ez[16] * Mz);
                        // dev_g[getIndexf(i, j, k, 18)] = dev_g[getIndexf(i, j, k, 17)] + geq[18] - geq[17] - (1.0 / 4.0) * (ex[18] * Mx + ez[18] * Mz);

                        if (q == 0)
                            dev_g[idxF] = geq[idx] - (dev_g[oppos_idxF] - geq[oppos_idx]);

                        else
                            dev_g[idxF] = geq[idx] - (dev_g[oppos_idxF] - geq[oppos_idx]) - N / 4.0;

                        // dev_h[getIndexf(i, j, k, 4)] = heq[4] - (dev_h[getIndexf(i, j, k, 3)] - heq[3]);
                        // dev_h[getIndexf(i, j, k, 8)] = heq[8] - (dev_h[getIndexf(i, j, k, 7)] - heq[7]) - N / 4.0;
                        // dev_h[getIndexf(i, j, k, 9)] = heq[9] - (dev_h[getIndexf(i, j, k, 10)] - heq[10]) - N / 4.0;
                        // dev_h[getIndexf(i, j, k, 16)] = heq[16] - (dev_h[getIndexf(i, j, k, 15)] - heq[15]) - N / 4.0;
                        // dev_h[getIndexf(i, j, k, 18)] = heq[18] - (dev_h[getIndexf(i, j, k, 17)] - heq[17]) - N / 4.0;
                    }
                }
                else
                {
                    for (int q = 0; q < 5; q++)
                    {

                        dev_g[ Domain::getIndex<Q>(domain, i, domain.ghostDomainMaxIdx[IDX_Y], k, wallBCMap.YPlus[q])] = dev_g[ Domain::getIndex<Q>(domain, i, domain.domainMaxIdx[IDX_Y], k, wallBCMap.YMinus[q])];
                        dev_f[ Domain::getIndex<Q>(domain, i, domain.ghostDomainMaxIdx[IDX_Y], k, wallBCMap.YPlus[q])] = dev_f[ Domain::getIndex<Q>(domain, i, domain.domainMaxIdx[IDX_Y], k, wallBCMap.YMinus[q])];
                    }
                }
            }

//             float R = (i - (*dev_x0 + 0.50)) * (i - (*dev_x0 + 0.50)) + (k - (*dev_z0 + 0.50)) * (k - (*dev_z0 + 0.50));

//             if (R < (*dev_r0 * *dev_r0))
//             {

//                 dev_uy[index] = *dev_u0;
//                 dev_ux[index] = 0;
//                 dev_uz[index] = 0;

//                 // dev_c[index] = 1.0L;
//                 dev_p[index] = 0.0L;

// #pragma unroll
//                 for (int q = 0; q < 19; q++)
//                 {
//                     // index = getIndex(i ,j ,k);
//                     float ga_wa = Equilibrium_new(0, *dev_u0, 0, q);
//                     geq[q] = 0.0 * wa[q] + ga_wa;

//                     // float eF = (4.0L * (dev_c[index] * (1.0L - dev_c[index])) * wa[q] * (ex[q] * dev_ni[index] + ey[q] * dev_nj[index] + ez[q] * dev_nk[index])) / *dev_w;
//                     heq[q] = dev_c[index] * (ga_wa + wa[q]);
//                 }

//                 dev_h[getIndexf(i, j, k, 0)] = heq[0];
//                 dev_g[getIndexf(i, j, k, 0)] = geq[0];

//                 CLIP_REAL Mx = 0;
//                 CLIP_REAL My = 0;
//                 CLIP_REAL Mz = 0;

// #pragma unroll
//                 for (int q = 0; q < 19; q++)
//                 {
//                     if (ey[0] == 0)
//                         Mx += ex[q] * (dev_g[getIndexf(i, j, k, q)] - geq[q]);

//                     if (ex[0] == 0)
//                         My += ey[q] * (dev_g[getIndexf(i, j, k, q)] - geq[q]);

//                     if (ex[0] == 0)
//                         Mz += ey[q] * (dev_g[getIndexf(i, j, k, q)] - geq[q]);
//                 }

//                 float Mx = (dev_g[getIndexf(i, j, k, 1)] - geq[1]) + (dev_g[getIndexf(i, j, k, 11)] - geq[11]) + (dev_g[getIndexf(i, j, k, 13)] - geq[13]) - (dev_g[getIndexf(i, j, k, 2)] - geq[2]) - (dev_g[getIndexf(i, j, k, 12)] - geq[12]) - (dev_g[getIndexf(i, j, k, 14)] - geq[14]);

//                 float Mz = (dev_g[getIndexf(i, j, k, 5)] - geq[5]) + (dev_g[getIndexf(i, j, k, 11)] - geq[11]) + (dev_g[getIndexf(i, j, k, 14)] - geq[14]) - (dev_g[getIndexf(i, j, k, 6)] - geq[6]) - (dev_g[getIndexf(i, j, k, 12)] - geq[12]) - (dev_g[getIndexf(i, j, k, 13)] - geq[13]);

//                 dev_g[getIndexf(i, j, k, 4)] = geq[4] + dev_g[getIndexf(i, j, k, 3)] - geq[3];
//                 dev_g[getIndexf(i, j, k, 8)] = dev_g[getIndexf(i, j, k, 7)] + geq[8] - geq[7] - (1.0 / 4.0) * (ex[8] * Mx + ez[8] * Mz);
//                 dev_g[getIndexf(i, j, k, 9)] = dev_g[getIndexf(i, j, k, 10)] + geq[9] - geq[10] - (1.0 / 4.0) * (ex[9] * Mx + ez[9] * Mz);
//                 dev_g[getIndexf(i, j, k, 16)] = dev_g[getIndexf(i, j, k, 15)] + geq[16] - geq[15] - (1.0 / 4.0) * (ex[16] * Mx + ez[16] * Mz);
//                 dev_g[getIndexf(i, j, k, 18)] = dev_g[getIndexf(i, j, k, 17)] + geq[18] - geq[17] - (1.0 / 4.0) * (ex[18] * Mx + ez[18] * Mz);

//                 float N = (dev_h[getIndexf(i, j, k, 1)] - heq[1]) + (dev_h[getIndexf(i, j, k, 2)] - heq[2]) + (dev_h[getIndexf(i, j, k, 5)] - heq[5]) + (dev_h[getIndexf(i, j, k, 6)] - heq[6]) + (dev_h[getIndexf(i, j, k, 11)] - heq[11]) +
//                           (dev_h[getIndexf(i, j, k, 12)] - heq[12]) + (dev_h[getIndexf(i, j, k, 13)] - heq[13]) +
//                           (dev_h[getIndexf(i, j, k, 14)] - heq[14]);

//                 dev_h[getIndexf(i, j, k, 4)] = heq[4] - (dev_h[getIndexf(i, j, k, 3)] - heq[3]);
//                 dev_h[getIndexf(i, j, k, 8)] = heq[8] - (dev_h[getIndexf(i, j, k, 7)] - heq[7]) - N / 4.0;
//                 dev_h[getIndexf(i, j, k, 9)] = heq[9] - (dev_h[getIndexf(i, j, k, 10)] - heq[10]) - N / 4.0;
//                 dev_h[getIndexf(i, j, k, 16)] = heq[16] - (dev_h[getIndexf(i, j, k, 15)] - heq[15]) - N / 4.0;
//                 dev_h[getIndexf(i, j, k, 18)] = heq[18] - (dev_h[getIndexf(i, j, k, 17)] - heq[17]) - N / 4.0;
//             }

//             else
//             {
//                 // dev_c[index] = 0.0L;
// #pragma unroll
//                 for (int q = 0; q < 5; q++)
//                 {
//                     dev_h[getIndexf(i, Ny_, k, FStop[q])] = dev_h[getIndexf(i, Ny_1, k, FStop[q + 5])];
//                     dev_g[getIndexf(i, Ny_, k, FStop[q])] = dev_g[getIndexf(i, Ny_1, k, FStop[q + 5])];
//                 }
//             }
        }
    }

    //////////////////////////////////////////////////////// end

    __global__ void kernelMirrorBoundary(const Domain::DomainInfo domain, const Boundary::BCTypeMap BCmap, CLIP_REAL *dev_a)
    {
        const CLIP_UINT i = THREAD_IDX_X;
        const CLIP_UINT j = THREAD_IDX_Y;
        const CLIP_UINT k = (DIM == 3) ? THREAD_IDX_Z : 0;

        if (Domain::isInside<DIM>(domain, i, j, k))
        {

            const CLIP_UINT idx = Domain::getIndex(domain, i, j, k);

            // XMIN
            if (Boundary::isMirrorType(BCmap.types[object::XMinus]) && i == domain.domainMinIdx[IDX_X])
            {
                const CLIP_UINT ghost = Domain::getIndex(domain, domain.ghostDomainMinIdx[IDX_X], j, k);
                dev_a[ghost] = dev_a[idx];
            }
            // XMAX
            if (Boundary::isMirrorType(BCmap.types[object::XPlus]) && i == domain.domainMaxIdx[IDX_X])
            {
                const CLIP_UINT ghost = Domain::getIndex(domain, domain.ghostDomainMaxIdx[IDX_X], j, k);
                dev_a[ghost] = dev_a[idx];
            }
            // YMIN
            if (Boundary::isMirrorType(BCmap.types[object::YMinus]) && j == domain.domainMinIdx[IDX_Y])
            {
                const CLIP_UINT ghost = Domain::getIndex(domain, i, domain.ghostDomainMinIdx[IDX_Y], k);
                dev_a[ghost] = dev_a[idx];
            }
            // YMAX
            if (Boundary::isMirrorType(BCmap.types[object::YPlus]) && j == domain.domainMaxIdx[IDX_Y])
            {
                const CLIP_UINT ghost = Domain::getIndex(domain, i, domain.ghostDomainMaxIdx[IDX_Y], k);
                dev_a[ghost] = dev_a[idx];
            }

#ifdef ENABLE_3D
            // ZMIN
            if (Boundary::isMirrorType(BCmap.types[object::ZMinus]) && k == domain.domainMinIdx[IDX_Z])
            {
                const CLIP_UINT ghost = Domain::getIndex(domain, i, j, domain.ghostDomainMinIdx[IDX_Z]);
                dev_a[ghost] = dev_a[idx];
            }
            // ZMAX
            if (Boundary::isMirrorType(BCmap.types[object::ZPlus]) && k == domain.domainMaxIdx[IDX_Z])
            {
                const CLIP_UINT ghost = Domain::getIndex(domain, i, j, domain.ghostDomainMaxIdx[IDX_Z]);
                dev_a[ghost] = dev_a[idx];
            }
#endif
        }
    }

    template <CLIP_UINT Q>
    void Solver::periodicBoundary(CLIP_REAL *dev_a, CLIP_REAL *dev_b)
    {
        if (m_boundary->isPeriodic)
            kernelPeriodicBoundary<Q><<<dimGrid, dimBlock>>>(m_info, m_BCMap, dev_a, dev_b);
    }

    template <CLIP_UINT Q, CLIP_UINT dof>
    void Solver::wallBoundary(CLIP_REAL *dev_a, CLIP_REAL *dev_a_post, CLIP_REAL *dev_b, CLIP_REAL *dev_b_post)
    {
        if (m_boundary->isWall)
            kernelHalfBounceBack<Q, dof, WMRT::wallBCMap><<<dimGrid, dimBlock>>>(m_info, m_BCMap, m_wallBCMap, dev_a, dev_a_post, dev_b, dev_b_post);
    }

    template <CLIP_UINT Q, CLIP_UINT dof>
    void Solver::slipWallBoundary(CLIP_REAL *dev_a, CLIP_REAL *dev_a_post, CLIP_REAL *dev_b, CLIP_REAL *dev_b_post)
    {
        if (m_boundary->isSlipWall)
            kernelHalfBounceBack<Q, dof, WMRT::slipWallBCMap><<<dimGrid, dimBlock>>>(m_info, m_BCMap, m_slipWallBCMap, dev_a, dev_a_post, dev_b, dev_b_post);
    }

    template <CLIP_UINT Q, CLIP_UINT dof>
    void Solver::freeConvectBoundary(CLIP_REAL *dev_vel, CLIP_REAL *dev_a, CLIP_REAL *dev_a_prev, CLIP_REAL *dev_b, CLIP_REAL *dev_b_prev)
    {
        if (m_boundary->isFreeConvect)
            kernelFreeConvect<Q, dof, WMRT::wallBCMap><<<dimGrid, dimBlock>>>(m_info, m_BCMap, m_wallBCMap, dev_vel, dev_a, dev_a_prev, dev_b, dev_b_prev);
    }

    template <CLIP_UINT Q, CLIP_UINT dof>
    void Solver::NeumannBoundary(CLIP_REAL *dev_a, CLIP_REAL *dev_b)
    {
        if (m_boundary->isNeumann)
            kernelNeumann<Q, dof, WMRT::wallBCMap><<<dimGrid, dimBlock>>>(m_info, m_BCMap, m_wallBCMap, dev_a, dev_b);
    }

    void Solver::mirrorBoundary(CLIP_REAL *dev_a)
    {
        if (m_boundary->isWall || m_boundary->isFreeConvect || m_boundary->isSlipWall || m_boundary->isNeumann)
            kernelMirrorBoundary<<<dimGrid, dimBlock>>>(m_info, m_BCMap, dev_a);
    }




    void Solver::velocityBoundary(CLIP_REAL *dev_c, CLIP_REAL *dev_f, CLIP_REAL *dev_g)
    {
        // if (m_boundary->isNeumann)
            JetBoundary<<<dimGrid, dimBlock>>>(m_info, m_geomPool, m_BCMap, m_velSet, m_slipWallBCMap, dev_c, dev_f, dev_g);


    }





    template void clip::Solver::periodicBoundary<9>(CLIP_REAL *, CLIP_REAL *);
    template void clip::Solver::periodicBoundary<19>(CLIP_REAL *, CLIP_REAL *);
    template void clip::Solver::periodicBoundary<1>(CLIP_REAL *, CLIP_REAL *);

    template void clip::Solver::wallBoundary<9, 3>(CLIP_REAL *, CLIP_REAL *, CLIP_REAL *, CLIP_REAL *);
    template void clip::Solver::wallBoundary<19, 5>(CLIP_REAL *, CLIP_REAL *, CLIP_REAL *, CLIP_REAL *);

    template void clip::Solver::slipWallBoundary<9, 3>(CLIP_REAL *, CLIP_REAL *, CLIP_REAL *, CLIP_REAL *);
    template void clip::Solver::slipWallBoundary<19, 5>(CLIP_REAL *, CLIP_REAL *, CLIP_REAL *, CLIP_REAL *);

    template void clip::Solver::freeConvectBoundary<9, 3>(CLIP_REAL *, CLIP_REAL *, CLIP_REAL *, CLIP_REAL *, CLIP_REAL *);
    template void clip::Solver::freeConvectBoundary<19, 5>(CLIP_REAL *, CLIP_REAL *, CLIP_REAL *, CLIP_REAL *, CLIP_REAL *);

    template void clip::Solver::NeumannBoundary<9, 3>(CLIP_REAL *, CLIP_REAL *);
    template void clip::Solver::NeumannBoundary<19, 5>(CLIP_REAL *, CLIP_REAL *);

}
