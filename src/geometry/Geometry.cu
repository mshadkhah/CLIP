#include "Geometry.cuh"
#include <fstream>
#include <sstream>
#include <algorithm>
#include <stdexcept>

namespace clip
{

Geometry::Geometry(const InputData& idata)
    : m_idata(&idata)
{
    Logger::Info("Reading geometries...");
    readGeometries(geometries);
    fillDeviceGeometry();
    print();
    Logger::Success("Successfully read " + std::to_string(geometries.size()) + " geometries.");
}

Geometry::~Geometry() {}

bool Geometry::readGeometries(std::vector<Entry>& geometries)
{
    geometryObjects = 0;
    geometries.clear();
    
    std::ifstream inputFile(m_idata->getConfig());
    if (!inputFile.is_open())
    {
        Logger::Error("Error opening config file: " + m_idata->getConfig());
    }

    std::string line;
    bool inGeometryList = false;
    bool inBlock = false;
    Entry current;

    while (std::getline(inputFile, line))
    {
        trim(line);
        if (line.empty() || line[0] == '#')
            continue;

        // Start of geometry
        if (!inGeometryList && line.find("geometry") != std::string::npos && line.find('=') != std::string::npos)
        {
            inGeometryList = true;
            continue;
        }

        // ✅ NEW: End of geometry list
        if (inGeometryList && line == "]")
        {
            break; // Stop reading once geometry block ends
        }

        if (inGeometryList)
        {
            if (line == "[")
                continue;
            if (line == "{")
            {
                inBlock = true;
                current = Entry{};
                continue;
            }
            if (line == "}" || line == "},")
            {
                inBlock = false;

                if (current.type == Type::Unknown)
                {
                    Logger::Error("Encountered a geometry with unknown type during parsing.");
                }

                geometries.push_back(current);
                geometryObjects++;
                continue;
            }

            if (inBlock)
            {
                std::size_t pos = line.find('=');
                if (pos != std::string::npos)
                {
                    std::string key = line.substr(0, pos);
                    std::string value = line.substr(pos + 1);
                    trim(key);
                    trim(value);
                    value.erase(std::remove(value.begin(), value.end(), '"'), value.end());

                    if (key == "type")
                        current.type = typeFromString(value);
                    else if (key == "center")
                    {
                        if (value.front() == '[') value.erase(0, 1);
                        if (value.back() == ']') value.pop_back();
                        std::stringstream ss(value);
                        std::string token;
                        int dim = 0;
                        while (std::getline(ss, token, ',') && dim < MAX_DIM)
                        {
                            trim(token);
                            current.center[dim++] = std::stod(token);
                        }
                    }
                    else if (key == "length")
                    {
                        if (value.front() == '[') value.erase(0, 1);
                        if (value.back() == ']') value.pop_back();
                        std::stringstream ss(value);
                        std::string token;
                        int dim = 0;
                        while (std::getline(ss, token, ',') && dim < MAX_DIM)
                        {
                            trim(token);
                            current.length[dim++] = std::stod(token);
                        }
                    }
                    else if (key == "radius")
                        current.radius = std::stod(value);
                    else if (key == "amplitude")
                        current.amplitude = std::stod(value);
                    else if (key == "id")
                        current.id = std::stoi(value);
                }
            }
        }
    }

    return !geometries.empty();
}

void Geometry::fillDeviceGeometry()
{
    m_deviceGeometry.numGeometries = geometries.size();
    for (size_t i = 0; i < geometries.size(); ++i)
    {
        m_deviceGeometry.type[i] = static_cast<int>(geometries[i].type);
        for (int d = 0; d < MAX_DIM; ++d)
        {
            m_deviceGeometry.center[i][d] = geometries[i].center[d];
            m_deviceGeometry.length[i][d] = geometries[i].length[d];
        }
        m_deviceGeometry.radius[i] = geometries[i].radius;
        m_deviceGeometry.amplitude[i] = geometries[i].amplitude;
        m_deviceGeometry.id[i] = geometries[i].id;
    }
}

std::string Geometry::toLower(const std::string& s)
{
    std::string result = s;
    std::transform(result.begin(), result.end(), result.begin(),
                   [](unsigned char c) { return std::tolower(c); });
    return result;
}

void Geometry::trim(std::string& s)
{
    size_t start = s.find_first_not_of(" \t");
    size_t end = s.find_last_not_of(" \t");
    if (start == std::string::npos)
        s.clear();
    else
        s = s.substr(start, end - start + 1);
}

Geometry::Type Geometry::typeFromString(const std::string& str)
{
    std::string lowerStr = toLower(str);

    if (lowerStr == "circle")
        return Type::Circle;
    if (lowerStr == "sphere")
        return Type::Sphere;
    if (lowerStr == "square")
        return Type::Square;
    if (lowerStr == "cube")
        return Type::Cube;
    if (lowerStr == "perturbation")
        return Type::Perturbation;

    return Type::Unknown;
}

std::string Geometry::typeToString(Type t) const
{
    switch (t)
    {
    case Type::Circle: return "Circle";
    case Type::Sphere: return "Sphere";
    case Type::Square: return "Square";
    case Type::Cube: return "Cube";
    case Type::Perturbation: return "Perturbation";
    default: return "Unknown";
    }
}

void Geometry::print() const
{
    std::cout << "\nParsed Geometries:\n";
    for (size_t i = 0; i < geometryObjects; ++i)
    {
        const auto& geo = geometries[i];
        std::cout << "  Geometry " << i << ":\n";
        std::cout << "    Type: " << typeToString(geo.type) << "\n";
        std::cout << "    ID: " << geo.id << "\n";
        std::cout << "    Center: [" << geo.center[0] << ", " << geo.center[1] << ", " << geo.center[2] << "]\n";
        std::cout << "    Radius: " << geo.radius << "\n";
        std::cout << "    Length: [" << geo.length[0] << ", " << geo.length[1] << ", " << geo.length[2] << "]\n";
        std::cout << "    Amplitude: " << geo.amplitude << "\n\n";
    }
}



} // namespace clip
